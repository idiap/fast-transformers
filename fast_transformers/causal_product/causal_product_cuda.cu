#include "hip/hip_runtime.h"
//
// Copyright (c) 2020 Idiap Research Institute, http://www.idiap.ch/
// Written by Angelos Katharopoulos <angelos.katharopoulos@idiap.ch>,
// Apoorv Vyas <avyas@idiap.ch>
//

//
// For modifications made inside namespace nvidia (authored by jdemouth):
//
// Copyright (c) 2021 NVIDIA CORPORATION. All rights reserved.
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy of
// this software and associated documentation files (the "Software"), to deal in
// the Software without restriction, including without limitation the rights to
// use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
// the Software, and to permit persons to whom the Software is furnished to do so,
// subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
// FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
// COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
// IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
// CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
//

#include <torch/extension.h>
#include <assert.h>
#include <stdio.h>

#define ENABLE_NVIDIA_OPTIMIZATIONS

#ifdef ENABLE_NVIDIA_OPTIMIZATIONS
namespace nvidia {

////////////////////////////////////////////////////////////////////////////////////////////////////

constexpr int THREADS_PER_WARP = 32;

////////////////////////////////////////////////////////////////////////////////////////////////////

constexpr int LOW_OCCUPANCY_THRESHOLD = 40; // TODO: Make it HW specific (like 1/2 SMs).

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ __host__ int div_up(int m, int n) {
  return (m + n-1) / n;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ __host__ int round_up(int m, int n) {
  return div_up(m, n) * n;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< typename T >
struct Lmha_params {

  // The output buffer. Dimensions [B, H, L, M].
  T *out;

  // The input Qs. Dimensions [B, H, L, E].
  const T *q;
  // The input Ks. Dimensions [B, H, L, E].
  const T *k;
  // The input Vs. Dimensions [B, H, L, M].
  const T *v;

  // The different dimensions.
  int B, L, H, E, M;

  // The strides for the different tensors.
  int q_stride_B, q_stride_H, q_stride_L;
  int k_stride_B, k_stride_H, k_stride_L;
  int v_stride_B, v_stride_H, v_stride_L;
  int o_stride_B, o_stride_H, o_stride_L;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, bool GO_BACKWARD, int WARPS, int COLS_PER_THREAD = 4 >
__global__ __launch_bounds__(WARPS * THREADS_PER_WARP)
void lmha_low_occupancy_kernel(Lmha_params<float> params) {

  // The number of threads per block.
  constexpr int THREADS_PER_BLOCK = WARPS * THREADS_PER_WARP;
  // The number of rows per thread.
  constexpr int ROWS_PER_THREAD = E / THREADS_PER_WARP;
  // The number of steps per iteration.
  constexpr int COLS_PER_ITER = WARPS * COLS_PER_THREAD;

  // Make sure E is a multiple of the warp size.
  static_assert(E % THREADS_PER_WARP == 0, "");

  // Shared memory to store V/O.
  __shared__ float smem_v[COLS_PER_ITER], smem_o[COLS_PER_ITER];
  // Shared memory buffer to performance the reductions.
  __shared__ float smem_reds[E * WARPS]; 

  // The sequence processed by that block.
  const int bi = blockIdx.z;
  // The head processed by that block.
  const int hi = blockIdx.y;
  // The hidden cell in the V/output buffers.
  const int vi = blockIdx.x;

  // The linear index of the thread.
  const int tidx = threadIdx.x;

  // Decompose the block in warp/lane.
  const int warp = tidx / THREADS_PER_WARP;
  const int lane = tidx % THREADS_PER_WARP;

  // The base offset loaded by the thread in Q and K.
  int offset_q = bi*params.q_stride_B + hi*params.q_stride_H + lane;
  int offset_k = bi*params.k_stride_B + hi*params.k_stride_H + lane;

  // If we walk backward, account for the extra offset.
  if( GO_BACKWARD ) {
    offset_q += (params.L-1)*params.q_stride_L;
    offset_k += (params.L-1)*params.k_stride_L;
  }

  // Position the warp at the beginning of the proper timestep.
  if( GO_BACKWARD ) {
    offset_q -= warp*COLS_PER_THREAD*params.q_stride_L;
    offset_k -= warp*COLS_PER_THREAD*params.k_stride_L;
  } else {
    offset_q += warp*COLS_PER_THREAD*params.q_stride_L;
    offset_k += warp*COLS_PER_THREAD*params.k_stride_L;
  }

  // Determine the base pointers for Q and K.
  const float *ptr_q = &params.q[offset_q];
  const float *ptr_k = &params.k[offset_k];

  // Is a given row valid?
  int valid_qk[ROWS_PER_THREAD];
  #pragma unroll
  for( int ii = 0; ii < ROWS_PER_THREAD; ++ii ) {
    valid_qk[ii] = lane + ii*THREADS_PER_WARP < params.E;
  }

  // The offset to the position loaded by the thread in V.
  int offset_v = bi*params.v_stride_B + hi*params.v_stride_H + vi;
  int offset_o = bi*params.o_stride_B + hi*params.o_stride_H + vi;

  // If we walk backward, account for the extra offset.
  if( GO_BACKWARD ) {
    offset_v += (params.L-1)*params.v_stride_L;
    offset_o += (params.L-1)*params.o_stride_L;
  }

  // We load/store a strided matrix of COLS_PER_ITER x OUTPUTS_PER_BLOCK.
  if( GO_BACKWARD ) {
    offset_v -= tidx*params.v_stride_L;
    offset_o -= tidx*params.o_stride_L;
  } else {
    offset_v += tidx*params.v_stride_L;
    offset_o += tidx*params.o_stride_L;
  }

  // Determine the base pointer for V.
  const float *ptr_v = &params.v[offset_v];
  // The output pointer. 
  float *ptr_o = &params.out[offset_o];

  // The running KVs.
  float running_kv[ROWS_PER_THREAD];
  #pragma unroll
  for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
    running_kv[ri] = 0.f;
  }

  // Iterate over the timesteps. TODO: Use params.loop_count!!!
  for( int iter = 0; iter < params.L; iter += COLS_PER_ITER ) {

    // Each thread loads a matrix of elements.
    float q[ROWS_PER_THREAD][COLS_PER_THREAD], k[ROWS_PER_THREAD][COLS_PER_THREAD];

    // Trigger the memory loads for Q and K.
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      #pragma unroll
      for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {

        // For Q/K, each warp loads from various timesteps. 
        int ti = iter + warp*COLS_PER_THREAD;
        if( GO_BACKWARD ) {
          ti = params.L - 1 - ti;
        }

        // Is it a valid access?
        int valid;
        if( GO_BACKWARD ) {
          valid = valid_qk[ri] && ti - ci >= 0;
        } else {
          valid = valid_qk[ri] && ti + ci < params.L;
        }

        // The extra offset to add.
        if( GO_BACKWARD ) {
          offset_q = ri*THREADS_PER_WARP - ci*params.q_stride_L;
          offset_k = ri*THREADS_PER_WARP - ci*params.k_stride_L;
        } else {
          offset_q = ri*THREADS_PER_WARP + ci*params.q_stride_L;
          offset_k = ri*THREADS_PER_WARP + ci*params.k_stride_L;
        }

        // Load Q/K if they are valid.
        q[ri][ci] = valid ? ptr_q[offset_q] : 0.f;
        k[ri][ci] = valid ? ptr_k[offset_k] : 0.f;
      }
    }

    // For the V tensor, we assign contiguous thread to different loads. So, ti is different.
    int ti = iter + tidx;
    if( GO_BACKWARD ) {
      ti = params.L - 1 - ti;
    }

    // Is it a valid access?
    int valid_vo = tidx < COLS_PER_ITER;
    if( GO_BACKWARD ) {
      valid_vo &= ti >= 0;
    } else {
      valid_vo &= ti < params.L;
    }

    // Trigger the loads for V. 
    float ldg_v = valid_vo ? *ptr_v : 0.f;

    // Move the load pointers.
    if( GO_BACKWARD ) {
      ptr_q -= COLS_PER_ITER*params.q_stride_L;
      ptr_k -= COLS_PER_ITER*params.k_stride_L;
      ptr_v -= COLS_PER_ITER*params.v_stride_L;
    } else {
      ptr_q += COLS_PER_ITER*params.q_stride_L;
      ptr_k += COLS_PER_ITER*params.k_stride_L;
      ptr_v += COLS_PER_ITER*params.v_stride_L;
    }

    // Store to shared memory.
    if( tidx < COLS_PER_ITER ) {
      smem_v[tidx] = ldg_v;
    }

    // Make sure V is in shared memory.
    __syncthreads();

    // Read V from shared memory.
    float v[COLS_PER_THREAD];
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      v[ci] = smem_v[warp*COLS_PER_THREAD + ci];
    }

    // Each thread computes local K*V products.
    float kv[ROWS_PER_THREAD][COLS_PER_THREAD];
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        kv[ri][ci] = 0.f;
      }
    }

    // Update the K*V^T product.
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      #pragma unroll
      for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
        kv[ri][ci] += k[ri][ci] * v[ci];
      }
    }

    // We must perform the prefix sums within the thread-block. Start with the thread.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      #pragma unroll
      for( int ci = 1; ci < COLS_PER_THREAD; ++ci ) {
        kv[ri][ci] += kv[ri][ci-1];
      }
    }

    // Store the partial sums to shared memory. Unless we have no inter-warp reduction to perform.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      smem_reds[warp*E + ri*THREADS_PER_WARP + lane] = kv[ri][COLS_PER_THREAD-1];
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // Each thread deals with one or more column(s) of the matrix.
    constexpr int SUMS_PER_THREAD = (E + THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
    #pragma unroll
    for( int ii = 0, idx = tidx; ii < SUMS_PER_THREAD; ++ii, idx += THREADS_PER_BLOCK ) {
      if( idx < E ) {
        float sum = smem_reds[idx];
        #pragma unroll
        for( int jj = 1; jj < WARPS; ++jj ) {
          smem_reds[idx + jj*E] = sum += smem_reds[idx + jj*E];
        }
      }
    }

    // Make sure the reductions are stored in shared memory.
    __syncthreads();

    // Each thread updates his partial products.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      float sum = running_kv[ri];
      if( warp > 0 ) {
        sum += smem_reds[(warp-1)*E + lane + ri*THREADS_PER_WARP];
      }
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        kv[ri][ci] += sum;
      }
    }

    // Compute the partial output values for that thread.
    float sum[COLS_PER_THREAD];
    #pragma unroll
    for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
      sum[ci] = q[0][ci] * kv[0][ci];
      #pragma unroll
      for( int ri = 1; ri < ROWS_PER_THREAD; ++ri ) {
        sum[ci] += q[ri][ci] * kv[ri][ci];
      }
    }

    // Run the parallel reductions inside the warp.
    #pragma unroll
    for( int mask = THREADS_PER_WARP / 2; mask >= 1; mask /= 2 ) {
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        sum[ci] += __shfl_xor_sync(uint32_t(-1), sum[ci], mask);
      }
    }

    // Store the final output to shared memory.
    if( lane == 0 ) {
      #pragma unroll
      for( int ci = 0; ci < COLS_PER_THREAD; ++ci ) {
        smem_o[warp*COLS_PER_THREAD + ci] = sum[ci];
      }
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // Store the output.
    if( valid_vo ) {
      *ptr_o = smem_o[tidx];
    }

    // Each thread updates his running kv.
    #pragma unroll
    for( int ri = 0; ri < ROWS_PER_THREAD; ++ri ) {
      running_kv[ri] += smem_reds[(WARPS-1)*E + lane + ri*THREADS_PER_WARP];
    }

    // Move to next location.
    if( GO_BACKWARD ) {
      ptr_o -= COLS_PER_ITER*params.o_stride_L;
    } else {
      ptr_o += COLS_PER_ITER*params.o_stride_L;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, bool GO_BACKWARD, int WARPS >
int lmha_low_occupancy_(const Lmha_params<float> &params) {

  // Make sure we are not going to launch an invalid grid.
  if( params.H > 65535 || params.B > 65535 ) {
    return 1;
  }

  // Prepare the grid and trigger the CUDA kernel.
  dim3 grid;
  grid.x = params.M;
  grid.y = params.H;
  grid.z = params.B;
  lmha_low_occupancy_kernel<E, GO_BACKWARD, WARPS><<<grid, WARPS*THREADS_PER_WARP>>>(params);
  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, bool GO_BACKWARD >
int lmha_low_occupancy_(const Lmha_params<float> &params, int blocks) {
         if( params.M * blocks >= 8*LOW_OCCUPANCY_THRESHOLD ) {
    return lmha_low_occupancy_<E, GO_BACKWARD,  4>(params);
  } else if( params.M * blocks >= 4*LOW_OCCUPANCY_THRESHOLD ) {
    return lmha_low_occupancy_<E, GO_BACKWARD,  8>(params);
  } else {
    return lmha_low_occupancy_<E, GO_BACKWARD, 16>(params);
  }
  return 1;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, typename Params >
static inline __device__ __host__ int smem_buffer_elts_(const Params &params) {
  int M = round_up(params.M, 4);
  return 2*E + 2*M;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, int THREADS_PER_HEAD, bool GO_BACKWARD >
__global__ 
void lmha_kernel(Lmha_params<float> params) {

  // Make sure E is a multiple of 4.
  static_assert(E % 4 == 0, "");

  // The amount of shared memory per buffer (2 buffers for double-buffering).
  const int smem_buffer_elts = smem_buffer_elts_<E>(params);
  // The M dimension for shared memory.
  const int M = round_up(params.M, 4);

  // Shared memory to store Q, K and V. Size is 2*smem_buffer_elts.
  extern __shared__ float smem_[];

  // The various shared memory buffers.
  float *smem_q = &smem_[0*E];
  float *smem_k = &smem_[1*E];
  float *smem_v = &smem_[2*E];
  float *smem_o = &smem_[2*E + M];

  // The index of the shared memory buffer (for double-buffering).
  int smem_curr = 0;

  // The sequence processed by that block.
  const int bi = blockIdx.y;
  // The head processed by that block.
  const int hi = blockIdx.x;

  // The linear index of the thread.
  const int tidx = threadIdx.x;

  // The offset to the position loaded by the thread in Q.
  int offset_q = bi*params.q_stride_B + hi*params.q_stride_H + tidx;
  // The offset to the position loaded by the thread in K.
  int offset_k = bi*params.k_stride_B + hi*params.k_stride_H + tidx;

  // If we walk backward, account for the extra offset.
  if( GO_BACKWARD ) {
    offset_q += (params.L-1)*params.q_stride_L;
    offset_k += (params.L-1)*params.k_stride_L;
  }

  // Determine the base pointers for Q and K.
  const float *ptr_q = &params.q[offset_q];
  const float *ptr_k = &params.k[offset_k];

  // The offset to the position loaded by the thread in V and O.
  int offset_v = bi*params.v_stride_B + hi*params.v_stride_H + tidx;
  int offset_o = bi*params.o_stride_B + hi*params.o_stride_H + tidx;

  // If we walk backward, account for the extra offset.
  if( GO_BACKWARD ) {
    offset_v += (params.L-1)*params.v_stride_L;
    offset_o += (params.L-1)*params.o_stride_L;
  }

  // Determine the base pointers for V.
  const float *ptr_v = &params.v[offset_v];

  // Is it an active Q/K thread?
  const int active_qk = tidx < params.E;

  // Trigger the memory loads for Q and K.
  float ldg_q = 0.f, ldg_k = 0.f;
  if( active_qk ) {
    ldg_q = *ptr_q;
    ldg_k = *ptr_k;
  }

  // Is it an active V thread?
  const int active_v = tidx < params.M;

  // Trigger the memory loads for V. 
  float ldg_v = 0.f;
  if( active_v ) {
    ldg_v = *ptr_v;
  }

  // Move the load pointers.
  if( GO_BACKWARD ) {
    ptr_q -= params.q_stride_L;
    ptr_k -= params.k_stride_L;
    ptr_v -= params.v_stride_L;
  } else {
    ptr_q += params.q_stride_L;
    ptr_k += params.k_stride_L;
    ptr_v += params.v_stride_L;
  }

  // The number of FLOAT4s per head.
  constexpr int FLOAT4s_PER_HEAD = E / 4;
  // The number of FLOAT4s per thread.
  constexpr int FLOAT4s_PER_THREAD = FLOAT4s_PER_HEAD / THREADS_PER_HEAD;

  // The storage for the K*V^T values.
  float4 kv[FLOAT4s_PER_THREAD]; 
  #pragma unroll
  for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
    kv[ii] = make_float4(0.f, 0.f, 0.f, 0.f);
  }

  // The output pointer.
  float *out_ptr = &params.out[offset_o];

  // Store to shared memory Q and K.
  if( tidx < E ) { 
    smem_q[smem_curr*smem_buffer_elts + tidx] = ldg_q; 
    smem_k[smem_curr*smem_buffer_elts + tidx] = ldg_k; 
  }

  // Store to shared memory V. All threads store valid values.
  if( tidx < M ) {
    smem_v[smem_curr*smem_buffer_elts + tidx] = ldg_v;
  }

  // The position of the thread in the V dimension.
  int vo = tidx / THREADS_PER_HEAD;
  int vi = tidx % THREADS_PER_HEAD;

  // Iterate over the timesteps.
  for( int ti = 0; ti < params.L; ++ti ) {

    // Is it the last iteration?
    int is_last = ti == params.L - 1;

    // Trigger the next loads for Q and K.
    if( !is_last && active_qk ) {
      ldg_q = *ptr_q;
      ldg_k = *ptr_k;
    }

    // Trigger the next loads for V.
    if( !is_last && active_v ) {
      ldg_v = *ptr_v;
    }

    // Move the load pointers.
    if( GO_BACKWARD ) {
      ptr_q -= params.q_stride_L;
      ptr_k -= params.k_stride_L;
      ptr_v -= params.v_stride_L;
    } else {
      ptr_q += params.q_stride_L;
      ptr_k += params.k_stride_L;
      ptr_v += params.v_stride_L;
    }

    // Make sure the data is in shared memory.
    __syncthreads();

    // Each thread loads 4 values from K.
    float4 k[FLOAT4s_PER_THREAD];
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      int ki = tidx % THREADS_PER_HEAD * 4 + ii * THREADS_PER_HEAD * 4;
      k[ii] = *reinterpret_cast<const float4*>(&smem_k[smem_curr*smem_buffer_elts + ki]);
    }

    // Each thread loads a single V value.
    float v = 0.f;
    if( vo < params.M ) {
      v = *reinterpret_cast<const float *>(&smem_v[smem_curr*smem_buffer_elts + vo]);
    }

    // Update the K*V^T product.
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      kv[ii].x += k[ii].x * v;
      kv[ii].y += k[ii].y * v;
      kv[ii].z += k[ii].z * v;
      kv[ii].w += k[ii].w * v;
    }

    // Load the Q values from shared memory.
    float4 q[FLOAT4s_PER_THREAD]; 
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      int qi = tidx % THREADS_PER_HEAD * 4 + ii * THREADS_PER_HEAD * 4;
      q[ii] = *reinterpret_cast<const float4*>(&smem_q[smem_curr*smem_buffer_elts + qi]);
    }

    // Compute the partial output value for that thread.
    float sum = 0.f;
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      sum += q[ii].x * kv[ii].x;
      sum += q[ii].y * kv[ii].y;
      sum += q[ii].z * kv[ii].z;
      sum += q[ii].w * kv[ii].w;
    }

    // Finalize the computation of the sum (if we have more than 1 thread per head).
    if( THREADS_PER_HEAD > 1 ) {

      // Finalize the sum for each head.
      #pragma unroll
      for( int mask = THREADS_PER_HEAD / 2; mask >= 1; mask /= 2 ) {
        sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
      }

      // Store to shared memory.
      if( vo < M && vi == 0 ) {
        smem_o[smem_curr*smem_buffer_elts + vo] = sum;
      }

      // Make sure the data is in shared memory.
      __syncthreads();

      // Active threads read the data to store.
      if( active_v ) {
        sum = smem_o[smem_curr*smem_buffer_elts + tidx];
      }

    } // THREADS_PER_HEAD > 1.

    // Store the output. All the threads are active.
    if( active_v ) {
      *out_ptr = sum;
    }

    // Move to next location.
    if( GO_BACKWARD ) {
      out_ptr -= params.o_stride_L;
    } else {
      out_ptr += params.o_stride_L;
    }

    // Move the shared memory buffer.
    smem_curr = (smem_curr + 1) % 2;

    // Store to shared memory for Q and K.
    if( !is_last && tidx < E ) {
      smem_q[smem_curr*smem_buffer_elts + tidx] = ldg_q;
      smem_k[smem_curr*smem_buffer_elts + tidx] = ldg_k;
    }

    // Store to shared memory for V.
    if( !is_last && tidx < M ) {
      smem_v[smem_curr*smem_buffer_elts + tidx] = ldg_v;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int E, int THREADS_PER_HEAD, bool GO_BACKWARD >
int lmha_(const Lmha_params<float> &params) {
  // The M dimension rounded up to 4.
  int M = round_up(params.M, 4);

  // The number of threads in the block.
  int block = round_up(max(E, M*THREADS_PER_HEAD), 32);
  if( block > 1024 || params.B > 65535 ) {
    return 1;
  }

  // Prepare the kernel.
  dim3 grid(params.H, params.B);
  size_t smem = smem_buffer_elts_<E>(params)*2*sizeof(float);
  lmha_kernel<E, THREADS_PER_HEAD, GO_BACKWARD><<<grid, block, smem>>>(params);
  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< bool GO_BACKWARD >
int lmha(const Lmha_params<float> &params) {
  int blocks = params.B * params.H;
  int res = 1;
  if( blocks < LOW_OCCUPANCY_THRESHOLD ) { 
           if( params.E <=  32 ) {
      res = lmha_low_occupancy_< 32, GO_BACKWARD>(params, blocks);
    } else if( params.E <=  64 ) {
      res = lmha_low_occupancy_< 64, GO_BACKWARD>(params, blocks);
    } else if( params.E <= 128 ) {
      res = lmha_low_occupancy_<128, GO_BACKWARD>(params, blocks);
    } else if( params.E <= 256 ) {
      res = lmha_low_occupancy_<256, GO_BACKWARD>(params, blocks);
    }
  } else {
           if( params.E <=  32 ) {
      res = lmha_< 32, 1, GO_BACKWARD>(params);
    } else if( params.E <=  48 ) {
      res = lmha_< 48, 1, GO_BACKWARD>(params);
    } else if( params.E <=  64 ) {
      res = lmha_< 64, 1, GO_BACKWARD>(params);
    } else if( params.E <= 128 ) {
      res = lmha_<128, 2, GO_BACKWARD>(params);
    } else if( params.E <= 256 ) {
      res = lmha_<256, 4, GO_BACKWARD>(params);
    }
  }
  return res;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< typename T >
inline void set_params(Lmha_params<T> &params,
                       const torch::Tensor q,
                       const torch::Tensor k,
                       const torch::Tensor v,
                       torch::Tensor       o) {

  // Define the pointers.
  params.out = o.data_ptr<T>();
  params.q   = q.data_ptr<T>();
  params.k   = k.data_ptr<T>();
  params.v   = v.data_ptr<T>();

  // Define the strides.
  params.q_stride_B = (int) q.stride(0);
  params.q_stride_H = (int) q.stride(1);
  params.q_stride_L = (int) q.stride(2);
  params.k_stride_B = (int) k.stride(0);
  params.k_stride_H = (int) k.stride(1);
  params.k_stride_L = (int) k.stride(2);
  params.v_stride_B = (int) v.stride(0);
  params.v_stride_H = (int) v.stride(1);
  params.v_stride_L = (int) v.stride(2);
  params.o_stride_B = (int) o.stride(0);
  params.o_stride_H = (int) o.stride(1);
  params.o_stride_L = (int) o.stride(2);

  // Extract the dimensions.
  int N = q.size(0);
  int H = q.size(1);
  int L = q.size(2);
  int E = q.size(3);
  int M = v.size(3);

  params.B = N;
  params.L = L;
  params.H  = H;
  params.E = E;
  params.M = M;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int lmha_fwd(const torch::Tensor queries,
             const torch::Tensor keys,
             const torch::Tensor values,
             torch::Tensor product) {

  // Make sure that we are using the correct GPU device
  torch::DeviceGuard _guard(queries.device());

  // Make sure the inner-most dimension of the tensors is packed.
  assert(queries.stride(3) == 1);
  assert(keys   .stride(3) == 1);
  assert(values .stride(3) == 1);
  assert(product.stride(3) == 1);

  // Extract the dimensions.
  int N = queries.size(0);
  int H = queries.size(1);
  int L = queries.size(2);
  int E = queries.size(3);
  int M = values.size (3);

  // The structure of params.
  Lmha_params<float> params;
  set_params(params, queries, keys, values, product);

  // Launch the kernel.
  return lmha<false>(params);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< typename T >
struct Lmha_bwd_params {

  // The output buffer for K. Dimensions [B, H, L, D].
  T *out_k;
  // The output buffer for V. Dimensions [B, H, L, D].
  T *out_v;

  // The input Qs. Dimensions [B, H, L, D].
  const T *q;
  // The input Ks. Dimensions [B, H, L, D].
  const T *k;
  // The input Vs. Dimensions [B, H, L, D].
  const T *v;
  // The input Gs. Dimensions [B, H, L, D].
  const T *g;

  // The dimensions.
  int B, L, H, M, E;

  // The strides for the input tensors.
  int q_stride_B, q_stride_L, q_stride_H;
  int k_stride_B, k_stride_L, k_stride_H;
  int v_stride_B, v_stride_L, v_stride_H;
  int g_stride_B, g_stride_L, g_stride_H;

  // The strides for the outputs.
  int out_k_stride_B, out_k_stride_L, out_k_stride_H;
  int out_v_stride_B, out_v_stride_L, out_v_stride_H;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int D, int THREADS_PER_HEAD >
__global__ __launch_bounds__(D*THREADS_PER_HEAD*2)
void lmha_bwd_kernel(Lmha_bwd_params<float> params) {

  // Make sure D is a multiple of 4.
  static_assert(D % 4 == 0, "");

  // The shared memory buffers.
  __shared__ struct Smem { float qg[2*D], kv[2*D], out_kv[2*D]; } smem_[2];

  // The index of the shared memory buffer (for double-buffering).
  int smem_curr = 0;

  // The sequence processed by that block.
  const int bi = blockIdx.y;
  // The head processed by that block.
  const int hi = blockIdx.x;

  // The linear index of the thread.
  const int tidx = threadIdx.x;

  // Split the threads into two slices.
  int so = tidx / (D*THREADS_PER_HEAD);
  int si = tidx % (D*THREADS_PER_HEAD);

  // The strides for B/L/H for the Q/G tensors.
  int qg_stride_B, qg_stride_L, qg_stride_H;
  if( so == 0 ) {
    qg_stride_B = params.q_stride_B;
    qg_stride_L = params.q_stride_L;
    qg_stride_H = params.q_stride_H;
  } else {
    qg_stride_B = params.g_stride_B;
    qg_stride_L = params.g_stride_L;
    qg_stride_H = params.g_stride_H;
  }

  // The strides for B/L/H for the K/V tensors.
  int kv_stride_B, kv_stride_L, kv_stride_H;
  if( so == 0 ) {
    kv_stride_B = params.k_stride_B;
    kv_stride_L = params.k_stride_L;
    kv_stride_H = params.k_stride_H;
  } else {
    kv_stride_B = params.v_stride_B;
    kv_stride_L = params.v_stride_L;
    kv_stride_H = params.v_stride_H;
  }

  // The hidden size.
  int hidden_size_per_head = 0;
  if( so == 0 ) {
    hidden_size_per_head = params.E;
  } else {
    hidden_size_per_head = params.M;
  }

  // Where to start reading from.
  int offset_qg = bi*qg_stride_B + hi*qg_stride_H + si;
  int offset_kv = bi*kv_stride_B + hi*kv_stride_H + si;

  // We walk backward, account for the extra offset.
  offset_qg += (params.L-1)*qg_stride_L;
  offset_kv += (params.L-1)*kv_stride_L;

  // Determine the base pointers for Q, K, V and G.
  const float *ptr_qg = &(so == 0 ? params.q : params.g)[offset_qg];
  const float *ptr_kv = &(so == 0 ? params.k : params.v)[offset_kv]; 

  // Is it an active thread?
  const int active = si < hidden_size_per_head;

  // Trigger the memory loads for Q, K, V and G.
  float ldg_qg = 0.f, ldg_kv = 0.f;
  if( active ) {
    ldg_qg = *ptr_qg;
    ldg_kv = *ptr_kv;
  }

  // Move the load pointers (backward).
  ptr_qg -= qg_stride_L;
  ptr_kv -= kv_stride_L;

  // The number of FLOAT4s per head.
  constexpr int FLOAT4s_PER_HEAD = D / 4;
  // The number of FLOAT4s per thread.
  constexpr int FLOAT4s_PER_THREAD = FLOAT4s_PER_HEAD / THREADS_PER_HEAD;

  // The storage for the G*Q^T or Q^T*G values.
  float4 gq[FLOAT4s_PER_THREAD]; 
  #pragma unroll
  for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
    gq[ii] = make_float4(0.f, 0.f, 0.f, 0.f);
  }

  // The strides for B/L/H for the K/V tensors.
  int out_kv_stride_B, out_kv_stride_L, out_kv_stride_H;
  if( so == 0 ) {
    out_kv_stride_B = params.out_k_stride_B;
    out_kv_stride_L = params.out_k_stride_L;
    out_kv_stride_H = params.out_k_stride_H;
  } else {
    out_kv_stride_B = params.out_v_stride_B;
    out_kv_stride_L = params.out_v_stride_L;
    out_kv_stride_H = params.out_v_stride_H;
  }

  // Where to start reading from.
  int offset_out_kv = bi*out_kv_stride_B + hi*out_kv_stride_H + si;

  // We walk backward, account for the extra offset.
  offset_out_kv += (params.L-1)*out_kv_stride_L;

  // The output pointer.
  float *ptr_out_kv = &(so == 0 ? params.out_k : params.out_v)[offset_out_kv];

  // Store to shared memory.
  if( si < D ) { 
    smem_[smem_curr].qg[so*D + si] = ldg_qg; 
    smem_[smem_curr].kv[so*D + si] = ldg_kv; 
  }

  // The position of the thread in the output dimension.
  int oo = si / THREADS_PER_HEAD % D;
  int oi = si % THREADS_PER_HEAD * 4;

  // Iterate over the timesteps.
  for( int ti = 0; ti < params.L; ++ti ) {

    // Is it the last iteration?
    int is_last = ti == params.L - 1;

    // Trigger the next loads.
    if( !is_last && active ) {
      ldg_qg = *ptr_qg;
      ldg_kv = *ptr_kv;
    }

    // Move the load pointers.
    ptr_qg -= qg_stride_L;
    ptr_kv -= kv_stride_L;

    // Make sure the data is in shared memory.
    __syncthreads();

    // Each thread loads 4 values from G or Q.
    float4 g[FLOAT4s_PER_THREAD];
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      float *smem_ptr = &smem_[smem_curr].qg[(so^1)*D + oi];
      g[ii] = *reinterpret_cast<const float4*>(&smem_ptr[ii*THREADS_PER_HEAD*4]);
    }

    // Each thread loads a single from Q or G value.
    float q = smem_[smem_curr].qg[so*D + oo];

    // Update the G*Q^T or Q*G^T product.
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      gq[ii].x += g[ii].x * q;
      gq[ii].y += g[ii].y * q;
      gq[ii].z += g[ii].z * q;
      gq[ii].w += g[ii].w * q;
    }

    // Load the V or K values from shared memory.
    float4 v[FLOAT4s_PER_THREAD]; 
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      float *smem_ptr = &smem_[smem_curr].kv[(so^1)*D + oi];
      v[ii] = *reinterpret_cast<const float4*>(&smem_ptr[ii*THREADS_PER_HEAD*4]);
    }

    // Compute the partial output value for that thread.
    float sum = 0.f;
    #pragma unroll
    for( int ii = 0; ii < FLOAT4s_PER_THREAD; ++ii ) {
      sum += v[ii].x * gq[ii].x;
      sum += v[ii].y * gq[ii].y;
      sum += v[ii].z * gq[ii].z;
      sum += v[ii].w * gq[ii].w;
    }

    // Finalize the computation of the sum (if we have more than 1 thread per head).
    if( THREADS_PER_HEAD > 1 ) {

      // Finalize the sum for each head.
      #pragma unroll
      for( int mask = THREADS_PER_HEAD / 2; mask >= 1; mask /= 2 ) {
        sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
      }

      // Store to shared memory.
      if( oi == 0 ) {
        smem_[smem_curr].out_kv[so*D + oo] = sum;
      }

      // Make sure the data is in shared memory.
      __syncthreads();

      // Active threads read the data to store.
      if( si < hidden_size_per_head ) {
        sum = smem_[smem_curr].out_kv[so*D + si];
      }

    } // THREADS_PER_HEAD > 1.

    // Store the output. All the threads are active.
    if( si < hidden_size_per_head ) {
      *ptr_out_kv = sum;
    }

    // Move to next location.
    ptr_out_kv -= out_kv_stride_L;

    // Move the shared memory buffer.
    smem_curr = (smem_curr + 1) % 2;

    // Store to shared memory for Q and K.
    if( !is_last && si < D ) {
      smem_[smem_curr].qg[so*D + si] = ldg_qg; 
      smem_[smem_curr].kv[so*D + si] = ldg_kv; 
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< int D, int THREADS_PER_HEAD >
int lmha_bwd_(const Lmha_bwd_params<float> &params) {
  int block = D*THREADS_PER_HEAD*2;
  if( block >= 1024 || params.B > 65535 ) {
    return 1;
  }
  dim3 grid(params.H, params.B);
  lmha_bwd_kernel<D, THREADS_PER_HEAD><<<grid, block>>>(params);
  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int lmha_bwd(const Lmha_bwd_params<float> &params) {
  int blocks = params.B * params.H;
  if( blocks < LOW_OCCUPANCY_THRESHOLD ) { 
    return 1;
  }

  int hidden_size_per_head = max(params.E, params.M);
  int res = 1;
  if( hidden_size_per_head <= 32 ) {
    res = lmha_bwd_< 32, 1>(params);
  } else if( hidden_size_per_head <= 64 ) {
    res = lmha_bwd_< 64, 1>(params);
  } else if( hidden_size_per_head <= 128 ) {
    res = lmha_bwd_<128, 2>(params);
  } else if( hidden_size_per_head <= 256 ) {
    res = lmha_bwd_<256, 4>(params);
  }
  return res;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int lmha_bwd(const torch::Tensor queries,
             const torch::Tensor keys,
             const torch::Tensor values,
             const torch::Tensor grad_out,
             torch::Tensor grad_queries,
             torch::Tensor grad_keys,
             torch::Tensor grad_values) {

  // Make sure that we are using the correct GPU device
  torch::DeviceGuard _guard(queries.device());

  // Make sure the inner-most dimension of the tensors is packed.
  assert(queries     .stride(3) == 1);
  assert(keys        .stride(3) == 1);
  assert(values      .stride(3) == 1);
  assert(grad_out    .stride(3) == 1);
  assert(grad_queries.stride(3) == 1);
  assert(grad_keys   .stride(3) == 1);
  assert(grad_values .stride(3) == 1);

  // Extract the dimensions.
  int N = queries.size(0);
  int H = queries.size(1);
  int L = queries.size(2);
  int E = queries.size(3);
  int M = values.size (3);

  // Gradient on Q.

  // The structure of params.
  Lmha_params<float> params;
  set_params(params, grad_out, values, keys, grad_queries);

  // Launch the kernel.
  int res = lmha<false>(params);
  if( res ) {
    return res;
  }

  // Gradient on K and V together.

  Lmha_bwd_params<float> bwd_params;
  bwd_params.out_k = grad_keys.data_ptr<float>();
  bwd_params.out_v = grad_values.data_ptr<float>();
  bwd_params.q = queries.data_ptr<float>();
  bwd_params.k = keys.data_ptr<float>();
  bwd_params.v = values.data_ptr<float>();
  bwd_params.g = grad_out.data_ptr<float>();

  bwd_params.B = N;
  bwd_params.L = L;
  bwd_params.H = H;
  bwd_params.E = E;
  bwd_params.M = M;

  bwd_params.q_stride_B = queries.stride(0);
  bwd_params.q_stride_H = queries.stride(1);
  bwd_params.q_stride_L = queries.stride(2);
  bwd_params.k_stride_B = keys.stride(0);
  bwd_params.k_stride_H = keys.stride(1);
  bwd_params.k_stride_L = keys.stride(2);
  bwd_params.v_stride_B = values.stride(0);
  bwd_params.v_stride_H = values.stride(1);
  bwd_params.v_stride_L = values.stride(2);
  bwd_params.g_stride_B = grad_out.stride(0);
  bwd_params.g_stride_H = grad_out.stride(1);
  bwd_params.g_stride_L = grad_out.stride(2);

  bwd_params.out_k_stride_B = grad_keys.stride(0);
  bwd_params.out_k_stride_H = grad_keys.stride(1);
  bwd_params.out_k_stride_L = grad_keys.stride(2);
  bwd_params.out_v_stride_B = grad_values.stride(0);
  bwd_params.out_v_stride_H = grad_values.stride(1);
  bwd_params.out_v_stride_L = grad_values.stride(2);

  // Try to run the fused kernel.
  int fallback = lmha_bwd(bwd_params);

  // If it failed, fallback on separate kernels for K and V.
  if( fallback ) {

    // Gradient on K.

    // Launch the kernel.
    set_params(params, values, grad_out, queries, grad_keys);
    res = lmha<true>(params);
    if( res ) {
      return res;
    }

    // Gradient on V.

    // Launch the kernel.
    set_params(params, keys, queries, grad_out, grad_values);
    return lmha<true>(params);
  }

  // It worked...
  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace nvidia
#endif // #ifdef ENABLE_NVIDIA_OPTIMIZATIONS

////////////////////////////////////////////////////////////////////////////////////////////////////

typedef torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> float_accessor;

__device__ void get_result(
    const float_accessor queries,
    const float_accessor keys,
    const float_accessor values,
    float_accessor kv,
    float_accessor result,
    const int n,
    const int h,
    const int e,
    const int m,
    const int L
) {
    for (int l=0; l<L; l++) {
        kv[n][h][e][m] += keys[n][h][l][e] * values[n][h][l][m];
        __syncthreads();
        float res = queries[n][h][l][e]*kv[n][h][e][m];
        atomicAdd(
            &result[n][h][l][m],
            res
        );
    }
}


__global__ void causal_dot_product_kernel(
    const float_accessor queries,
    const float_accessor keys,
    const float_accessor values,
    float_accessor kv,
    float_accessor result,
    const int N,
    const int H,
    const int L,
    const int E,
    const int M,
    const int E_per_block,
    const int blocks_per_sequence,
    const int T,
    const int l_offset
) {
    const int sequence_index = blockIdx.x / blocks_per_sequence;
    int n = sequence_index / H;
    int h = sequence_index % H;

    int e_local = threadIdx.x / M;
    int e_start = ((blockIdx.x % blocks_per_sequence) * E_per_block);
    int e = e_start + e_local;
    int m = threadIdx.x % M;

    // Load the shared memory for KV
    const int shared_kv_size = E_per_block * M;
    extern __shared__ float shared_mem[];
    float* shared_kv = shared_mem;
    float* shared_results = shared_mem + shared_kv_size;
    float* shared_values = shared_results + M;
    float* shared_keys = shared_values + M*T;
    float* shared_queries = shared_keys + E_per_block*T;

    if (threadIdx.x < M) {
        shared_results[threadIdx.x] = 0.0;
    }

    int t_end = (T + l_offset) <= L ? T : L - l_offset;
    for (int i = threadIdx.x; i < (t_end*M); i += blockDim.x)
    {
        int t = int(i / M) + l_offset;
        int d = i % M;
        shared_values[i] = values[n][h][t][d];
    }
    for (int i = threadIdx.x; i < (t_end*E_per_block); i += blockDim.x)
    {
        int t = int(i / E_per_block) + l_offset;
        int d = (i % E_per_block) + e_start;
        if (d < E) {
            shared_keys[i] = keys[n][h][t][d];
            shared_queries[i] = queries[n][h][t][d];
        }
    }
    __syncthreads();
    if ((n >= N) || (e >= E)) {
        return;
    }
    shared_kv[threadIdx.x] = kv[n][h][e][m];
    for (int t=0; t<t_end; t++) {
        int l = t + l_offset;
        shared_kv[e_local*M + m] += shared_keys[t*E_per_block + e_local] * shared_values[t*M + m];
        __syncthreads();
        float res = shared_queries[t*E_per_block + e_local] * shared_kv[e_local*M + m];
        atomicAdd(
            &shared_results[m],
            res
        );
        __syncthreads();
        if (threadIdx.x < M) {
            float r1 = shared_results[threadIdx.x];
            atomicAdd(
                &result[n][h][l][m],
                r1
            );
            shared_results[threadIdx.x] = 0.0;
        }
    }
    __syncthreads();
    kv[n][h][e][m] = shared_kv[e_local*M + m];
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void causal_dot_product_(const torch::Tensor queries,
                         const torch::Tensor keys,
                         const torch::Tensor values,
                         torch::Tensor product) {
    // Make sure that we are using the correct GPU device
    torch::DeviceGuard _guard(queries.device());

    int N = queries.size(0);
    int H = queries.size(1);
    int L = queries.size(2);
    int E = queries.size(3);
    int M = values.size(3);

    auto kv = torch::zeros({N, H, E, M}, queries.options());

    int threads = 1024;

    // Shared mem max size is 48KB
    int MUL_PER_BLOCK = min(threads, E*M);
    // make sure that MUL_PER_BLOCK is divisible by M;
    MUL_PER_BLOCK = int(MUL_PER_BLOCK / M) *  M;
    threads = MUL_PER_BLOCK;
    const int blocks_per_sequence = ((E*M) + threads -1) / threads;

    const int E_per_block = MUL_PER_BLOCK / M;
    int blocks  = N*H*blocks_per_sequence;
    int shared_mem_const = (E_per_block + 1)*M;
    int shared_mem_per_time = (M + 2*E_per_block);
    const int T = int(((12 * 1024) - shared_mem_const) / shared_mem_per_time);
    const int shared_mem_forward = ((T*shared_mem_per_time) + shared_mem_const) * sizeof(float);

    for (int l_offset=0; l_offset < L; l_offset += T) {
     causal_dot_product_kernel
            <<<blocks, MUL_PER_BLOCK, shared_mem_forward>>>(
            queries.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            keys.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            values.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            kv.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            product.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            N, H, L, E, M, E_per_block, blocks_per_sequence, T, l_offset
        );
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void causal_dot_product(const torch::Tensor queries,
                        const torch::Tensor keys,
                        const torch::Tensor values,
                        torch::Tensor product) {
#ifdef ENABLE_NVIDIA_OPTIMIZATIONS
  int fallback = nvidia::lmha_fwd(queries, keys, values, product);
#else
  int fallback = 1;
#endif
  if( fallback ) {
    causal_dot_product_(queries, keys, values, product);
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// we need shared memory to store
// Forward direction
// keys, values, gradout
// kv, results
// Backward direction
// queries, gradout, values
// kv_backwards, results
// Shared memory usage
// Forward
// keys: E*T, (values, gradout): M_per_block*T, kv:E*M_per_block, results:E
// Backward
// queries: E*T, (values, gradout): M_per_block*T, kv:E*M_per_block, results:E
// Total memory:
__global__ void causal_dot_backward_query_key_kernel(
    const float_accessor queries,
    const float_accessor keys,
    const float_accessor values,
    const float_accessor grad_out,
    float_accessor kv,
    float_accessor kv_backwards,
    float_accessor grad_queries,
    float_accessor grad_keys,
    int N,
    int H,
    int L,
    int E,
    int M,
    const int M_per_block,
    const int blocks_per_sequence,
    const int T,
    const int l_offset
) {
    const int sequence_index = blockIdx.x / blocks_per_sequence;
    int n = sequence_index / H;
    int h = sequence_index % H;

    int m_local = threadIdx.x / E;
    int m_start = ((blockIdx.x % blocks_per_sequence)*M_per_block);
    int m = m_start + m_local;
    int e = threadIdx.x % E;

    // Load the shared memory
    // Forward memory
    // keys: E*T, (values, gradout): M_per_block*T, kv:E*M_per_block, results:E
    // Backward memory
    // queries: E*T, (values, gradout): M_per_block*T, kv:E*M_per_block, results:E
    // Load the shared memory for KV
    extern __shared__ float shared_mem[];
    const int shared_kv_size = M_per_block * E;
    float* shared_kv = shared_mem;
    float* shared_kv_bw = shared_mem + shared_kv_size;
    float* shared_results = shared_kv_bw + shared_kv_size;
    float* shared_results_bw = shared_results + E;
    float* shared_keys = shared_results_bw + E;
    float* shared_values = shared_keys + E*T;
    float* shared_gradout = shared_values + M_per_block*T;
    float* shared_queries_bw = shared_gradout + M_per_block*T;
    float* shared_values_bw = shared_queries_bw + E*T;
    float* shared_gradout_bw = shared_values_bw + M_per_block*T;

    if (threadIdx.x < E) {
        shared_results[threadIdx.x] = 0.0;
        shared_results_bw[threadIdx.x] = 0.0;
    }

    int t_end = (T + l_offset) <= L ? T : (L - l_offset);
    for (int i = threadIdx.x; i < (t_end*M_per_block); i += blockDim.x)
    {
        int t = int(i / M_per_block) + l_offset;
        int t_bw = L - t - 1;
        int d = (i % M_per_block) + m_start;
        if (d < M) {
            shared_values[i] = values[n][h][t][d];
            shared_gradout[i] = grad_out[n][h][t][d];
            shared_values_bw[i] = values[n][h][t_bw][d];
            shared_gradout_bw[i] = grad_out[n][h][t_bw][d];
        }
    }
    for (int i = threadIdx.x; i < (t_end*E); i += blockDim.x)
    {
        int t = int(i / E) + l_offset;
        int t_bw = L - t - 1;
        int d = (i % E);
        shared_keys[i] = keys[n][h][t][d];
        shared_queries_bw[i] = queries[n][h][t_bw][d];
    }
    __syncthreads();

    if ((n >= N) || (m >= M)) {
        return;
    }

    shared_kv[threadIdx.x] = kv[n][h][e][m];
    shared_kv_bw[threadIdx.x] = kv_backwards[n][h][e][m];

    for (int t=0; t<t_end; t++) {
        int l = t + l_offset;
        int l_b = L - l -1;
        shared_kv[m_local*E + e] += shared_keys[t*E + e] * shared_values[t*M_per_block + m_local];
        shared_kv_bw[m_local*E + e] += shared_queries_bw[t*E + e] * shared_gradout_bw[t*M_per_block + m_local];
        __syncthreads();
        float res = shared_gradout[t*M_per_block + m_local] * shared_kv[m_local*E + e];
        float res_bw = shared_values_bw[t*M_per_block + m_local] * shared_kv_bw[m_local*E + e];
        atomicAdd(
            &shared_results[e],
            res
        );
        atomicAdd(
            &shared_results_bw[e],
            res_bw
        );
        __syncthreads();
        if (threadIdx.x < E) {
            float rq = shared_results[threadIdx.x];
            float rk = shared_results_bw[threadIdx.x];
            atomicAdd(
                &grad_queries[n][h][l][e],
                rq
            );
            atomicAdd(
                &grad_keys[n][h][l_b][e],
                rk
            );
            shared_results[threadIdx.x] = 0.0;
            shared_results_bw[threadIdx.x] = 0.0;
        }
    }
    __syncthreads();
    kv[n][h][e][m] = shared_kv[m_local*E + e];
    kv_backwards[n][h][e][m] = shared_kv_bw[m_local*E + e];
}


__global__ void causal_dot_backward_value_kernel(
    const float_accessor queries,
    const float_accessor keys,
    const float_accessor values,
    const float_accessor grad_out,
    float_accessor kv,
    float_accessor grad_keys,
    float_accessor grad_values,
    int N,
    int H,
    int L,
    int E,
    int M,
    int E_per_block,
    int blocks_per_sequence,
    int T,
    int l_offset
) {
    const int sequence_index = blockIdx.x / blocks_per_sequence;
    int n = sequence_index / H;
    int h = sequence_index % H;

    int e_local = threadIdx.x / M;
    int e_start = ((blockIdx.x % blocks_per_sequence) * E_per_block);
    int e = e_start + e_local;
    int m = threadIdx.x % M;

    // Load the shared memory for KV
    const int shared_kv_size = E_per_block * M;
    extern __shared__ float shared_mem[];
    float* shared_kv = shared_mem;
    float* shared_results = shared_mem + shared_kv_size;
    float* shared_gradout = shared_results + M;
    float* shared_keys = shared_gradout + M*T;
    float* shared_queries = shared_keys + E_per_block*T;

    if (threadIdx.x < M) {
        shared_results[threadIdx.x] = 0.0;
    }

    int t_end = (T + l_offset) <= L ? T : L - l_offset;
    for (int i = threadIdx.x; i < (t_end*M); i += blockDim.x)
    {
        int t = int(i / M) + l_offset;
        int t_bw = L - 1 - t;
        int d = i % M;
        shared_gradout[i] = grad_out[n][h][t_bw][d];
    }
    for (int i = threadIdx.x; i < (t_end*E_per_block); i += blockDim.x)
    {
        int t = int(i / E_per_block) + l_offset;
        int t_bw = L - 1 - t;
        int d = (i % E_per_block) + e_start;
        if (d < E) {
            shared_keys[i] = keys[n][h][t_bw][d];
            shared_queries[i] = queries[n][h][t_bw][d];
        }
    }
    __syncthreads();

    if ((n >= N) || (e >= E)){
        return;
    }

    shared_kv[threadIdx.x] = kv[n][h][e][m];
    for (int t=0; t<t_end; t++) {
        int l = t + l_offset;
        int l_b = L - l -1;
        shared_kv[e_local*M + m] += shared_queries[t*E_per_block + e_local] * shared_gradout[t*M + m];
        __syncthreads();
        float res = shared_keys[t*E_per_block + e_local] * shared_kv[e_local*M + m];
        atomicAdd(
            &shared_results[m],
            res
        );
        __syncthreads();
        if (threadIdx.x < M) {
            float r1 = shared_results[threadIdx.x];
            atomicAdd(
                &grad_values[n][h][l_b][m],
                r1
            );
            shared_results[threadIdx.x] = 0.0;
        }
    }
    __syncthreads();
    kv[n][h][e][m] = shared_kv[e_local*M + m];
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void causal_dot_backward_(const torch::Tensor queries,
                          const torch::Tensor keys,
                          const torch::Tensor values,
                          const torch::Tensor grad_out,
                          torch::Tensor grad_queries,
                          torch::Tensor grad_keys,
                          torch::Tensor grad_values) {

    // Make sure that we are using the correct GPU device
    torch::DeviceGuard _guard(queries.device());

    int N = queries.size(0);
    int H = queries.size(1);
    int L = queries.size(2);
    int E = queries.size(3);
    int M = values.size(3);

    auto kv = torch::zeros({N, H, E, M}, queries.options());
    auto kv_backward = torch::zeros({N, H, E, M}, queries.options());

    const int threads = 1024;
    int MUL_PER_BLOCK = min(threads, E*M);
    // make sure that MUL_PER_BLOCK is divisible by M;
    MUL_PER_BLOCK = int(MUL_PER_BLOCK / E) *  E;
    const int blocks_per_sequence = ((E*M) + MUL_PER_BLOCK -1) / MUL_PER_BLOCK;
    const int M_per_block = MUL_PER_BLOCK / E;
    int blocks  = N*H*blocks_per_sequence;

    // Forward memory
    // keys: E*T, (values, gradout): M_per_block*T, kv:E*M_per_block, results:E
    // Backward memory
    // queries: E*T, (values, gradout): M_per_block*T, kv:E*M_per_block, results:E
    // Total memory
    // 2*((E + 2*M_per_block)*T + (E+1)*M_per_block)
    int shared_mem_const = 2*E*(1+M_per_block);
    int shared_mem_per_time = 2*(E + 2*M_per_block);
    int T = int(((12 * 1024) - shared_mem_const) / shared_mem_per_time);
    const int shared_mem_qk_backward = ((T*shared_mem_per_time) + shared_mem_const) * sizeof(float);
    for (int l_offset=0; l_offset < L; l_offset += T) {
        causal_dot_backward_query_key_kernel
            <<<blocks, MUL_PER_BLOCK, shared_mem_qk_backward>>>(
            queries.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            keys.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            values.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            grad_out.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            kv.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            kv_backward.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            grad_queries.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            grad_keys.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            N, H, L, E, M, M_per_block, blocks_per_sequence, T, l_offset
        );
    }

    int MPB = min(threads, E*M);
    // make sure that MUL_PER_BLOCK is divisible by M;
    MPB = int(MPB / M) *  M;
    const int blocks_per_sequence_value = ((E*M) + MPB - 1)/ MPB;
    const int E_per_block = MPB / M;
    const int blocks_value  = N*H*blocks_per_sequence_value;

    shared_mem_const = (E_per_block + 1)*M;
    shared_mem_per_time = (M + 2*E_per_block);
    T = int(((12 * 1024) - shared_mem_const) / shared_mem_per_time);
    const int shared_mem_v_backward = ((T*shared_mem_per_time) + shared_mem_const) * sizeof(float);
    kv.zero_();
    for (int l_offset=0; l_offset < L; l_offset += T) {
        causal_dot_backward_value_kernel
            <<<blocks_value, MPB, shared_mem_v_backward>>>(
            queries.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            keys.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            values.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            grad_out.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            kv.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            grad_keys.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            grad_values.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
            N, H, L, E, M, E_per_block, blocks_per_sequence_value, T, l_offset
        );
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void causal_dot_backward(const torch::Tensor queries,
                         const torch::Tensor keys,
                         const torch::Tensor values,
                         const torch::Tensor grad_out,
                         torch::Tensor grad_queries,
                         torch::Tensor grad_keys,
                         torch::Tensor grad_values) {
#ifdef ENABLE_NVIDIA_OPTIMIZATIONS
  int fallback = nvidia::lmha_bwd(queries,
                                  keys,
                                  values,
                                  grad_out,
                                  grad_queries,
                                  grad_keys,
                                  grad_values);
#else
  int fallback = 1;
#endif
  if( fallback ) {
    causal_dot_backward_(queries, keys, values, grad_out, grad_queries, grad_keys, grad_values);
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def(
        "causal_dot_product",
        &causal_dot_product,
        "Compute the weighted sum of values but attending only to previous "
        "values."
    );
    m.def(
        "causal_dot_backward",
        &causal_dot_backward,
        "Compute the gradients for the causal dot product."
    );
}

